#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>
#include <Windows.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
	for(int j = 0; j < 1000000; j++)
		c[i] = a[i] * b[i];
}

double get_cpu_time(){
    FILETIME a,b,c,d;
    if (GetProcessTimes(GetCurrentProcess(),&a,&b,&c,&d) != 0){
        //  Returns total user time.
        //  Can be tweaked to include kernel times as well.
        return
            (double)(d.dwLowDateTime |
            ((unsigned long long)d.dwHighDateTime << 32)) * 0.0000001;
    }else{
        //  Handle error
        return 0;
    }
}

int main()
{
    int arraySize = 1024;
    int* a = new int[arraySize];
    int* b = new int[arraySize];
	int* c = new int[arraySize];

	for(auto i = 0; i < arraySize; i++)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	getch();

    // Add vectors in parallel.	
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	
	double cpu0  = get_cpu_time();

	for(int i = 0; i < arraySize; i++)
	{
		for(int j = 0; j < 1000000; j++)
			c[i] = a[i] * b[i];
	}


	double cpu1  = get_cpu_time();
	printf("CPU time: %3.1f ms\n", (cpu1 - cpu0) * 1000);


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	delete[] a;
	delete[] b;
	delete[] c;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	hipEvent_t start, stop;
	hipEventCreate(&start, 0);
	hipEventCreate(&stop, 0);	

	hipEventRecord(start, 0);

    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("GPU time: %3.1f ms\n" , elapsedTime);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
